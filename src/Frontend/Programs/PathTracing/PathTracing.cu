#include "hip/hip_runtime.h"
#include "UniUtils/ConversionUtils.h"
#include "glm/glm.hpp"
#include <optix_device.h>

#include "Device/Common.h"
#include "Device/Pdf.h"
#include "Device/Sample.h"
#include "Device/Scene.h"

#include "PathTracingLaunchParams.h"

using namespace EasyRender;
using namespace EasyRender::Device;
using namespace EasyRender::Programs::PathTracing;

__constant__ float EPSILON = 1e-3;

enum STRATEGY
{
    UPT = 0,
    NEE = 1,
    MIS = 2,
    STRATEGY_MAX
};

__constant__ STRATEGY strategy = MIS;

static __forceinline__ __device__ float UptMisWeight(float uptPdf, float neePdf)
{
    assert(uptPdf + neePdf > 0);
    //printf("%f\n", uptPdf / (uptPdf + neePdf));
    return uptPdf / (uptPdf + neePdf);
}

extern "C" __constant__ Programs::PathTracing::LaunchParams param;

/* PG id - 0 */
extern "C" __global__ void __raygen__RenderFrame()
{

    auto idx_x = optixGetLaunchIndex().x, idx_y = optixGetLaunchIndex().y;
    auto idx = (std::size_t)optixGetLaunchDimensions().x * idx_y + idx_x;
    Payload prd;

    /* Generate random seed */
    prd.seed = tea<4>(idx, param.frameID);
    prd.depth = 0;
    prd.done = false;
    prd.radiance = { 0, 0, 0 };
    prd.throughput = { 1, 1, 1 };
    prd.lastTraceTerm = 1.f;
    prd.rayPos = param.camera.pos;
    prd.rayDir = PinholeGenerateRay({ idx_x, idx_y }, param.fbSize,
                                    param.camera, prd.seed);

    std::uint32_t u0, u1;
    PackPointer(&prd, u0, u1);

    float RR_rate = 0.8;
    while (true)
    {

        optixTrace(param.traversable, UniUtils::ToFloat3(prd.rayPos),
                   UniUtils::ToFloat3(prd.rayDir), EPSILON, 1e30, 0,
                   OptixVisibilityMask(255), OPTIX_RAY_FLAG_DISABLE_ANYHIT,
                   RADIANCE_TYPE, RAY_TYPE_COUNT, RADIANCE_TYPE, u0, u1);
        if (prd.done)
        {
            break;
        }

        if (strategy != UPT)
        {
            LightSample ls;
            SampleAreaLightPos(param.areaLightCount, param.areaLights, ls,
                               prd.seed);

            /* Visibility test */
            glm::vec3 visRay = ls.pos - prd.rayPos;
            float dist = glm::length(visRay);
            visRay /= dist;
            std::uint32_t vis = 0;
            optixTrace(param.traversable, UniUtils::ToFloat3(prd.rayPos),
                       UniUtils::ToFloat3(visRay), EPSILON,
                       dist * (1 - EPSILON), 0, OptixVisibilityMask(255),
                       // For shadow rays: skip any/closest hit shaders and
                       // terminate on first intersection with anything. The
                       // miss shader is used to mark if the light was visible.
                       OPTIX_RAY_FLAG_DISABLE_ANYHIT |
                           OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT |
                           OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT,
                       SHADOW_TYPE, RAY_TYPE_COUNT, SHADOW_TYPE, vis);
            if (vis > 0)
            {
                float cosTheta1 = glm::dot(ls.N, visRay);
                float cosTheta2 = glm::dot(prd.lastNormal, visRay);
                auto &lt = param.areaLights[ls.areaLightID];
                if ((lt.twoSided || cosTheta1 < 0) && cosTheta2 > 0)
                {
                    float neePdf = ls.pdf * dist * dist / fabsf(cosTheta1);
                    float uptPdf = (strategy == NEE) ? 0 : RECIP_2PI;
                    prd.radiance += prd.throughput * cosTheta2 * lt.L * 2.0f /
                                    neePdf * (1 - UptMisWeight(uptPdf, neePdf));
                }
            }
        }

        if (prd.depth > 5)
        {
            if (rnd(prd.seed) > RR_rate)
            {
                prd.throughput = { 0, 0, 0 };
                break;
            }
            prd.throughput /= RR_rate;
        }
    }

    glm::dvec4 thisFrame = { prd.radiance.x, prd.radiance.y, prd.radiance.z,
                             0xFF };
    int frameID = param.frameID;
    if (frameID == 0)
        param.radianceBuffer[idx] = thisFrame;
    else
    {
        glm::dvec4 lastFrame = param.radianceBuffer[idx];
        param.radianceBuffer[idx] =
            lastFrame * double(frameID / (frameID + 1.0f)) +
            thisFrame * double(1.0f / (frameID + 1.0f));
    }
    param.colorBuffer[idx] =
        glm::clamp(param.radianceBuffer[idx], 0.f, 1.f) * 255.0f;
}

/* PG id - 1 */
extern "C" __global__ void __miss__radiance()
{
    auto *prd = GetPRD<Payload>();

    prd->radiance +=
        prd->throughput *
        reinterpret_cast<MissData *>(optixGetSbtDataPointer())->bg_color;
    prd->done = true;
}

/* PG id - 2 */
extern "C" __global__ void __miss__shadow()
{
    /* Visibility = true */
    optixSetPayload_0(114514);
}

/* PG id - 3 */
extern "C" __global__ void __closesthit__radiance()
{
    auto *prd = GetPRD<Payload>();
    HitData *mat = reinterpret_cast<HitData *>(optixGetSbtDataPointer());
    prd->depth += 1;

    uint32_t primIdx = optixGetPrimitiveIndex();
    glm::ivec3 indices = mat->indices[optixGetPrimitiveIndex()];
    glm::vec3 N = BarycentricByIndices(mat->normals, indices,
                                       optixGetTriangleBarycentrics());
    N = glm::normalize(N);

    glm::vec3 hitPos = GetHitPosition();

    /* Hit light */
    if (mat->areaLightID < INVALID_INDEX)
    {
        auto &lt = param.areaLights[mat->areaLightID];
        if ((prd->depth == 1 || (strategy != NEE)) &&
            (lt.twoSided || glm::dot(N, prd->rayDir) < 0))
        {
            /* Directly hit light, NEE does not participate*/
            float neePdf = 0.f;
            if (strategy != UPT && prd->depth > 1)
            {
                LightSample ls;
                ls.pos = hitPos;
                ls.areaLightID = mat->areaLightID;
                PdfAreaLightPos(param.areaLightCount, param.areaLights, primIdx,
                                ls);
                float dist = optixGetRayTmax();
                neePdf = ls.pdf * dist * dist / fabs(-glm::dot(N, prd->rayDir));
            }

            float uptPdf = RECIP_2PI;
            prd->radiance += prd->throughput * lt.L *
                             fabs(glm::dot(prd->lastNormal, prd->rayDir)) *
                             2.0f * UptMisWeight(uptPdf, neePdf);
        }
        prd->done = true;
        return;
    }
    if (prd->depth >= 25)
    {
        prd->throughput = { 0, 0, 0 };
        prd->done = true;
        return;
    }
    if (glm::dot(N, prd->rayDir) > 0)
        N = -N;

    float pdf;
    glm::vec3 rayDir = SampleUniformHemisphere(N, pdf, prd->seed);

    auto cosWeight = fmaxf(0.f, glm::dot(rayDir, N));
    prd->throughput *= mat->Kd / PI; // pdf = 1 / 2pi, albedo = kd / pi
    prd->throughput *= prd->lastTraceTerm;
    prd->lastTraceTerm = cosWeight / pdf;
    prd->rayPos = hitPos;
    prd->rayDir = rayDir;
    prd->lastNormal = N;

    return;
}

/* PG id - 4 */
extern "C" __global__ void __anyhit__shadow() {}