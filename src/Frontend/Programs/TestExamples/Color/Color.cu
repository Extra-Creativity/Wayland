#include "hip/hip_runtime.h"
#include <optix_device.h>

#include "Device/Camera.h"
#include "ColorLaunchParams.h"
#include "UniUtils/ConversionUtils.h"

using namespace EasyRender;

extern "C" __constant__ ColorLaunchParams param;

enum
{
    RADIANCE_TYPE = 0,
    RAY_TYPE_COUNT
};

template<typename T>
__device__ void PackPointer(T &data, std::uint32_t &u0, std::uint32_t &u1)
{
    auto ptr = reinterpret_cast<std::uintptr_t>(&data);
    u1 = ptr, u0 = ptr >> 32;
}

template<typename T>
__device__ T &UnpackPointer(std::uint32_t u0, std::uint32_t u1)
{
    return *reinterpret_cast<T *>(std::uintptr_t{ u0 } << 32 | u1);
}

/*! helper function that creates a semi-random color from an ID */
inline __device__ float3 randomColor(int i)
{
    int r = unsigned(i) * 13 * 17 + 0x234235;
    int g = unsigned(i) * 7 * 3 * 5 + 0x773477;
    int b = unsigned(i) * 11 * 19 + 0x223766;
    return { (r & 255) / 255.f, (g & 255) / 255.f, (b & 255) / 255.f };
}

extern "C" __global__ void __raygen__RenderFrame()
{
    auto idx_x = optixGetLaunchIndex().x, idx_y = optixGetLaunchIndex().y;
    //  scale to [-1, 1], row goes vertically.
    float xPos = 2.f * idx_x / optixGetLaunchDimensions().x - 1;
    float yPos = 2.f * idx_y / optixGetLaunchDimensions().y - 1;

    float3 result{ 0.8, 0.8, 0.8 };
    std::uint32_t u0, u1;
    PackPointer(result, u0, u1);

    // Normally we need a scale to shift the ray direction, here just omit it.
    glm::vec3 rayDir =
        glm::normalize(param.camera.lookAt + xPos * param.camera.right +
                       yPos * param.camera.up);

    rayDir = PinholeGenerateRay({ idx_x, idx_y }, param.fbSize, param.camera);

    optixTrace(param.traversable, UniUtils::ToFloat3(param.camera.pos),
               UniUtils::ToFloat3(rayDir), 1e-5, 1e30, 0, 255,
               OPTIX_RAY_FLAG_DISABLE_ANYHIT, RADIANCE_TYPE, RAY_TYPE_COUNT,
               RADIANCE_TYPE, u0, u1);

    auto idx = (std::size_t)optixGetLaunchDimensions().x * idx_y + idx_x;
    param.colorBuffer[idx].r = result.x * 255;
    param.colorBuffer[idx].g = result.y * 255;
    param.colorBuffer[idx].b = result.z * 255;
    param.colorBuffer[idx].a = 0xFF;
}

extern "C" __global__ void __miss__radiance()
{
    auto &result =
        UnpackPointer<float3>(optixGetPayload_0(), optixGetPayload_1());
    result = { 0.0, 0.0, 0.0 };
}

extern "C" __global__ void __closesthit__radiance()
{
    auto &result =
        UnpackPointer<float3>(optixGetPayload_0(), optixGetPayload_1());
    int primID = optixGetPrimitiveIndex();
    result = randomColor(primID);
}

extern "C" __global__ void __anyhit__radiance()
{ /*! for this simple example, this will remain empty */
}