#include "hip/hip_runtime.h"
#include <optix_device.h>

#include "DepthLaunchParams.h"
#include "Device/Camera.h"
#include "UniUtils/ConversionUtils.h"

using namespace EasyRender;

extern "C" __constant__ DepthLaunchParams param;

enum
{
    RADIANCE_TYPE = 0,
    RAY_TYPE_COUNT
};

template<typename T>
__device__ void PackPointer(T &data, std::uint32_t &u0, std::uint32_t &u1)
{
    auto ptr = reinterpret_cast<std::uintptr_t>(&data);
    u1 = ptr, u0 = ptr >> 32;
}

template<typename T>
__device__ T &UnpackPointer(std::uint32_t u0, std::uint32_t u1)
{
    return *reinterpret_cast<T *>(std::uintptr_t{ u0 } << 32 | u1);
}

extern "C" __global__ void __raygen__RenderFrame()
{
    auto idx_x = optixGetLaunchIndex().x, idx_y = optixGetLaunchIndex().y;

    float result;
    std::uint32_t u0, u1;
    PackPointer(result, u0, u1);

    glm::vec3 rayDir = PinholeGenerateRay(
        { idx_x, idx_y }, param.fbSize, param.camera);

    optixTrace(param.traversable, UniUtils::ToFloat3(param.camera.pos),
               UniUtils::ToFloat3(rayDir), 1e-5, 1e30, 0, 255,
               OPTIX_RAY_FLAG_DISABLE_ANYHIT, RADIANCE_TYPE, RAY_TYPE_COUNT,
               RADIANCE_TYPE, u0, u1);

    auto idx = (std::size_t)optixGetLaunchDimensions().x * idx_y + idx_x;

    // if (result>0) printf("%f\n", result);
    param.depthBuffer[idx] = result;
    if (param.frameID > 1)
    {
        float t = (result - param.minDepth) / (param.maxDepth - param.minDepth);
        unsigned int color;
        if (t < 0)
            color = 0;
        else
            color = 50 + (1 - t) * 200;
        param.colorBuffer[idx].r = color;
        param.colorBuffer[idx].g = color;
        param.colorBuffer[idx].b = color;
        param.colorBuffer[idx].a = 0xFF;
    }
}

extern "C" __global__ void __miss__radiance()
{
    auto &result =
        UnpackPointer<float>(optixGetPayload_0(), optixGetPayload_1());
    result = -1;
}

extern "C" __global__ void __closesthit__radiance()
{
    auto &result =
        UnpackPointer<float>(optixGetPayload_0(), optixGetPayload_1());
    result = optixGetRayTmax();
}

extern "C" __global__ void __anyhit__radiance()
{ /*! for this simple example, this will remain empty */
}