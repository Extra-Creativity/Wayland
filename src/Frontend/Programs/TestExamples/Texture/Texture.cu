#include "hip/hip_runtime.h"
#include <optix_device.h>

#include "Device/Camera.h"
#include "Device/Common.h"

#include "TextureLaunchParams.h"
#include "UniUtils/ConversionUtils.h"

using namespace EasyRender;
using namespace EasyRender::Device;
using namespace EasyRender::Programs::Texture;

extern "C" __constant__ Programs::Texture::LaunchParams param;

enum
{
    RADIANCE_TYPE = 0,
    RAY_TYPE_COUNT
};

template<typename T>
__device__ void PackPointer(T &data, std::uint32_t &u0, std::uint32_t &u1)
{
    auto ptr = reinterpret_cast<std::uintptr_t>(&data);
    u1 = ptr, u0 = ptr >> 32;
}

template<typename T>
__device__ T &UnpackPointer(std::uint32_t u0, std::uint32_t u1)
{
    return *reinterpret_cast<T *>(std::uintptr_t{ u0 } << 32 | u1);
}

extern "C" __global__ void __raygen__RenderFrame()
{
    auto idx_x = optixGetLaunchIndex().x, idx_y = optixGetLaunchIndex().y;

    glm::vec3 result{ 0.0, 0.0, 0.0 };
    std::uint32_t u0, u1;
    PackPointer(result, u0, u1);

    // Normally we need a scale to shift the ray direction, here just omit it.
    glm::vec3 rayDir =
        PinholeGenerateRay({ idx_x, idx_y }, param.fbSize, param.camera);

    optixTrace(param.traversable, UniUtils::ToFloat3(param.camera.pos),
               UniUtils::ToFloat3(rayDir), 1e-5, 1e30, 0, 255,
               OPTIX_RAY_FLAG_DISABLE_ANYHIT, RADIANCE_TYPE, RAY_TYPE_COUNT,
               RADIANCE_TYPE, u0, u1);

    auto idx = (std::size_t)optixGetLaunchDimensions().x * idx_y + idx_x;
    param.colorBuffer[idx].r = result.x * 255;
    param.colorBuffer[idx].g = result.y * 255;
    param.colorBuffer[idx].b = result.z * 255;
    param.colorBuffer[idx].a = 0xFF;
}

extern "C" __global__ void __miss__radiance()
{
    auto &result =
        UnpackPointer<glm::vec3>(optixGetPayload_0(), optixGetPayload_1());
    result = reinterpret_cast<MissData *>(optixGetSbtDataPointer())->bg_color;
}

extern "C" __global__ void __closesthit__radiance()
{
    auto &result =
        UnpackPointer<glm::vec3>(optixGetPayload_0(), optixGetPayload_1());

    const int primID = optixGetPrimitiveIndex();
    glm::vec2 barcentrics = { optixGetTriangleBarycentrics().x,
                              optixGetTriangleBarycentrics().y };

    HitData *mat = reinterpret_cast<HitData *>(optixGetSbtDataPointer());
    glm::ivec3 indices = mat->index[primID];

    if (mat->hasTexture)
    {
        glm::vec2 UV[4] = { { 0, 0 }, { 1, 0 }, { 1, 1 }, { 0, 1 } };
        glm::vec2 tc =
            BarycentricByIndices(UV, indices, barcentrics);
         glm::vec4 texColor = UniUtils::ToVec4<glm::vec4>(
             tex2D<float4>(mat->texture, tc.x, tc.y));
         result = texColor;
    }
    else
    {
        result = mat->Kd;
    }
}

extern "C" __global__ void __anyhit__radiance()
{ /*! for this simple example, this will remain empty */
}